#include <stdio.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>

// 16x16=64 threads per block
#define BLOCKDIMx 16   
#define BLOCKDIMy 16   

/**
 *
 * DFIRE distances
 *
 **/
__device__ static unsigned int dist_to_bins[50] = {
         1,  1,  1,  2,  3,  4,  5,  6,  7,  8,
         9, 10, 11, 12, 13, 14, 14, 15, 15, 16,
        16, 17, 17, 18, 18, 19, 19, 20, 20, 21,
        21, 22, 22, 23, 23, 24, 24, 25, 25, 26,
        26, 27, 27, 28, 28, 29, 29, 30, 30, 31};


// error checking macro
#define cudaCheck(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)




__shared__ double receptor[BLOCKDIMx][3];
__shared__ double ligand[BLOCKDIMy][3];
//__device__ unsigned int nindex = 0;
//__device__ int ninterface = -1;
//__device__ double *penergy; 

void __global__ compute_distance(double ** rec_array, double ** lig_array, unsigned int rec_len, unsigned int lig_len,
                                 double * dist){

   unsigned int i, j, x, y;
   x = threadIdx.x;
   y = threadIdx.y;

   i = blockIdx.y *blockDim.y +y;
   j = blockIdx.x *blockDim.x +x;
   
   if( (i<rec_len) && (j<lig_len))
   {
   	if(x==0) {
       		receptor[y][0] = rec_array[i][0];
		receptor[y][1] = rec_array[i][1];
		receptor[y][2] = rec_array[i][2];
   	}
	__syncthreads();

   	if(y==0){
		ligand[x][0] = lig_array[j][0];
		ligand[x][1] = lig_array[j][1];
		ligand[x][2] = lig_array[j][2];
   	}
	__syncthreads();

   	double sub1, sub2, sub3;
 	unsigned int index = i*lig_len +j; 

	sub1 = receptor[y][0] - ligand[x][0];
        sub2 = receptor[y][1] - ligand[x][1];
        sub3 = receptor[y][2] - ligand[x][2];
        dist[index] = sub1*sub1 + sub2*sub2 + sub3*sub3;

   }
}

__shared__ unsigned long receptor_obj[BLOCKDIMx];
__shared__ unsigned long ligand_obj[BLOCKDIMy];
void __global__ compute_neighbours( unsigned long * rec_obj, unsigned long * lig_obj, unsigned int rec_len, unsigned int lig_len, double * dist,
                                    unsigned int * interface_receptor,  unsigned int * interface_ligand, int *interface_len, double interface_cutoff, 
				    unsigned int * array, int *index_len){

   unsigned int i, j, x, y;
   x = threadIdx.x;
   y = threadIdx.y;

   i = blockIdx.y *blockDim.y +y;
   j = blockIdx.x *blockDim.x +x;
   if( (i<rec_len) && (j<lig_len))
   {
        if(x==0) {
                receptor_obj[y] = rec_obj[i];
        }
        __syncthreads();

        if(y==0){
                ligand_obj[x] = lig_obj[j];
        }
        __syncthreads();

	unsigned int sqrt_dist;
        unsigned int index = i*lig_len +j;
        unsigned long dfire_bin;    
        unsigned int prev_index;  

 	if (dist[index] <= 225.) {
		sqrt_dist = (sqrt(dist[index])*2.0 - 1.0);
		if (sqrt_dist < interface_cutoff){
                        prev_index = atomicAdd(interface_len, 1);
                	//atomicInc(&(*interface_len), rec_len);
                	interface_receptor[prev_index] = i;
                	interface_ligand[prev_index] = j;
		}
		dfire_bin = dist_to_bins[sqrt_dist] -1;
		//atomicInc(indexes_len, rec_len*lig_len);
		array[atomicAdd(index_len, 1)] = receptor_obj[y]*3360 + ligand_obj[x]*20 + dfire_bin;                
        }
    }
}

  
__device__ double compute_partial_energy(double * dfire_en_array, unsigned int * array, int N , unsigned int en_len){

   __shared__ double senergy[BLOCKDIMx*BLOCKDIMy];
   unsigned int i, x;
   x  =  threadIdx.x;
   i = blockDim.x*blockIdx.x +x;

   senergy[x] = 0.0;
   unsigned int index;
   double energyi = 0.0;
   while (i < N){
        index = array[i];
        if ( index < en_len )  energyi = dfire_en_array[index];
        senergy[x] += energyi;
        i += gridDim.x*blockDim.x;
   }

   for( unsigned int k = blockDim.x/2; k>0; k>>=1){
        __syncthreads();
        if(x < k)
            senergy[x] += senergy[x+k];
   }
   return senergy[x];
}

__device__ double compute_total_energy(volatile double * energy){

   __shared__ double tot_energy;
   unsigned int i, x;
   x  =  threadIdx.x;
   i = blockDim.x*blockIdx.x +x;
   unsigned int lastblock = 64; //gridDim.x-1;
   //if(blockIdx.x == lastblock){
  	 if (x == 0){
                unsigned int k = 0;
                while(k < lastblock)
        	tot_energy = energy[k++];
               
   	}
   	__syncthreads();
 /*
   	if (i < lastblock){
        	double penergy = energy[i];
        //	atomicAdd(&tot_energy, penergy);
       // tot_energy += energy[i];
   	}
*/
   return tot_energy;  

}


__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__global__ void compute_energy(double * dfire_en_array, unsigned int * array, int N, unsigned int en_len, volatile double* energy){
 
    double partial_energy = compute_partial_energy( dfire_en_array, array, N, en_len);

    if (threadIdx.x == 0) {
        energy[blockIdx.x] = partial_energy;

        __threadfence();

        unsigned int value = atomicInc(&count, gridDim.x);
        isLastBlockDone = (value == (gridDim.x - 1));
    }
    __syncthreads();

    if (isLastBlockDone) {
        if(blockIdx.x == (gridDim.x -1)){
        double total_energy = compute_total_energy(energy);
        if (threadIdx.x == 0) {
            energy[0] = total_energy;
            count = 0;
        }
    }
    }

}



/*
 *  Acceleratable code using GPU or OpenACC etc.
 *
 */


void compute_acc(double ** rec_array, double ** lig_array, unsigned int rec_len, unsigned int lig_len,
                    unsigned long * rec_obj, unsigned long * lig_obj, unsigned int ** interface_receptor,
                    unsigned int ** interface_ligand, double interface_cutoff, unsigned int *interface_len,
		    double * dfire_en_array, double *energy){
 
   unsigned int en_len = 564480;
   int * h_index_len = new int; 
   int * h_interface_len = new int; 
   double * h_energy = new double;
   double ** d_rec_array,  ** d_lig_array,* d_dist;
   unsigned long * d_rec_obj, * d_lig_obj;
   unsigned int  * d_interface_receptor, * d_interface_ligand;
   unsigned int  * d_array;
   int * d_interface_len, *d_index_len;
   double * d_dfire_en_array;
   double * d_energy;
   unsigned int nblocks = 64;
    
   size_t rec_bytes = rec_len*sizeof(double);
   size_t lig_bytes = lig_len*sizeof(double);
   size_t rec_lbytes = rec_len*sizeof(unsigned long);
   size_t lig_lbytes = lig_len*sizeof(unsigned long);
   
   unsigned int rl_len = rec_len*lig_len;
   size_t rl_ibytes = rl_len*sizeof(unsigned int);
   size_t rl_bytes = rl_len*sizeof(double);
   printf("rl: %d \t rec: %d \t lig: %d \n", rl_len, rec_len, lig_len);  
   (*interface_receptor) = (unsigned int *) malloc(rl_ibytes);
   (*interface_ligand) = (unsigned int *)malloc(rl_ibytes);
 
   unsigned int * array = (unsigned int *) malloc(rl_ibytes);

   hipMalloc(&d_interface_len, sizeof(int));
   cudaCheck("Memory allocation for d_interface_len is failed ");
   hipMalloc(&d_index_len, sizeof(int));
   cudaCheck("Memory allocation for d_index_len is failed ");

   hipMalloc(&d_rec_array, rec_bytes);
   cudaCheck("Memory allocation for d_rec_array is failed ");
   hipMalloc(&d_lig_array, lig_bytes);
   cudaCheck("Memory allocation for d_lig_array is failed ");
   hipMalloc(&d_dist, rl_bytes);
   cudaCheck("Memory allocation for distance is failed ");
   hipMalloc(&d_rec_obj, rec_lbytes);
   cudaCheck("Memory allocation for d_rec_obj is failed ");
   hipMalloc(&d_lig_obj, lig_lbytes);
   cudaCheck("Memory allocation for d_lig_obj is failed ");
   hipMalloc(&d_interface_receptor, rl_ibytes);
   cudaCheck("Memory allocation for d_interface_receptor is failed ");
   hipMalloc(&d_interface_ligand, rl_ibytes);
   cudaCheck("Memory allocation for d_interface_ligand is failed ");
   hipMalloc(&d_array, rl_ibytes);
   cudaCheck("Memory allocation for d_array is failed ");
   hipMemset(d_array, 0, rl_ibytes);
   cudaCheck("Memory set for d_array is failed ");
   hipMemset(d_index_len, 0, sizeof(int));
   cudaCheck("Memory set for d_index_len is failed ");
   hipMemset(d_interface_len, 0, sizeof(int));
   cudaCheck("Memory set for d_interface_len is failed ");
   
   unsigned int en_bytes = en_len*sizeof(double);  
   hipMalloc(&d_dfire_en_array, en_bytes);
   cudaCheck("Memory allocation for d_dfire_en_array is failed ");
   hipMalloc(&d_energy, nblocks*sizeof(double));
   cudaCheck("Memory allocation for d_energy is failed ");
  
   hipMemset(d_energy, 0, nblocks*sizeof(double));
   cudaCheck("Memory set for d_energy is failed ");
   
   unsigned int nstreams = 3;
   hipStream_t stream[nstreams];
   for( unsigned int i =0; i < nstreams; ++i)
        hipStreamCreate(&stream[i]);
   
   hipMemcpyAsync(d_rec_array, rec_array, rec_bytes, hipMemcpyHostToDevice, stream[0]);
   cudaCheck("Data transfer from H2D for d_rec_array is failed ");
   hipMemcpyAsync(d_lig_array, lig_array, lig_bytes, hipMemcpyHostToDevice, stream[0]);
   cudaCheck("Data transfer from H2D for d_rec_array is failed ");
   
   hipMemcpyAsync(d_rec_obj, rec_obj, rec_lbytes, hipMemcpyHostToDevice, stream[1]);
   cudaCheck("Data transfer from H2D for d_rec_array is failed ");
   hipMemcpyAsync(d_lig_obj, lig_obj, lig_lbytes, hipMemcpyHostToDevice, stream[1]);
   cudaCheck("Data transfer from H2D for d_lig_obj is failed ");
   
   hipMemcpyAsync(d_dfire_en_array, dfire_en_array, en_bytes, hipMemcpyHostToDevice, stream[2]);
   cudaCheck("Data transfer from H2D for d_dfire_en_array is failed ");

   const dim3 blockSize(BLOCKDIMx, BLOCKDIMy);
   int GRIDDIMx = (BLOCKDIMx +rec_len -1)/BLOCKDIMx;
   int GRIDDIMy = (BLOCKDIMy +lig_len -1)/BLOCKDIMy;
   const dim3 gridSize( GRIDDIMx, GRIDDIMy);

  //  int nblocks = GRIDDIMx*GRIDDIMy;
  // cudaProfilerStart();

   compute_distance<<< gridSize, blockSize, 0, stream[0] >>>(d_rec_array, d_lig_array, rec_len, lig_len, d_dist);
   cudaCheck(" compute_dist kernel launching is failed ");
 
   hipStreamSynchronize(stream[0]);    //--- Make sure the completion of the distance computation
   
   compute_neighbours<<< gridSize, blockSize, 0, stream[1] >>>( d_rec_obj, d_lig_obj, rec_len, lig_len, d_dist,
                         d_interface_receptor, d_interface_ligand, d_interface_len, interface_cutoff, d_array, d_index_len);
   cudaCheck(" compute_neighbour kernel launching is failed ");
   
   hipMemcpyAsync(h_index_len, d_index_len, sizeof(int), hipMemcpyDeviceToHost, stream[1]);
   cudaCheck("Data transfer from D2H for index_len is failed ");

   int index_len = *h_index_len; 
   const unsigned int nthreads = BLOCKDIMx *BLOCKDIMy;
 //  unsigned int nblocks = ceil((index_len +nthreads -1)/nthreads);
   const dim3 blockSize1D(nthreads);
   const dim3 gridSize1D(nblocks);
   
   hipMemcpyAsync(h_interface_len, d_interface_len, sizeof(int), hipMemcpyDeviceToHost, stream[1]);
   cudaCheck("Data transfer from D2H for interface_len is failed ");
   printf("h_interfac::  %d \n", (*h_interface_len));

   hipMemcpyAsync((*interface_receptor), d_interface_receptor, rl_ibytes, hipMemcpyDeviceToHost, stream[1]);
   cudaCheck("Data transfer from D2H for interface_receptor is failed ");
   hipMemcpyAsync((*interface_ligand), d_interface_ligand, rl_ibytes, hipMemcpyDeviceToHost, stream[1]);
   cudaCheck("Data transfer from D2H for interface_ligand is failed ");
  
   hipStreamSynchronize(stream[1]);    //--- Make sure the completion of the neighbor list computation
   compute_energy<<< gridSize1D, blockSize1D, 0, stream[2]>>>(d_dfire_en_array, d_array, index_len, en_len, d_energy);
   cudaCheck(" calculate_energy kernel launching is failed ");

   hipMemcpyAsync(h_energy, d_energy, sizeof(double), hipMemcpyDeviceToHost, stream[2]);
   cudaCheck("Data transfer from D2H for energy is failed ");
   printf("energy: %f \n", (*h_energy));   

  // cudaProfilerStop();
   
   (*energy) = (*h_energy);
   (*interface_len) = (*h_interface_len);
   
   //Free Device memory
   hipFree(d_rec_array);
   hipFree(d_lig_array);
   hipFree(d_dist);
   hipFree(d_rec_obj);
   hipFree(d_lig_obj);
   hipFree(d_interface_receptor);
   hipFree(d_interface_ligand);
   hipFree(d_interface_len);
   hipFree(d_index_len);
   hipFree(d_array);
   hipFree(d_dfire_en_array);
   hipFree(d_energy);

   //---------------------------------------------------------

}



